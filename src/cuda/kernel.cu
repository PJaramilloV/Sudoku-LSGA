#include "hip/hip_runtime.h"
#include "kernel.cuh"

/*__global__ void vec_sum(int *a, int *b, int *c, int n){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}*/

int somehow_get_another() {
  return 0;
}


__global__ void lsga_col_kernel(unsigned char* grid, unsigned char* occupancy, unsigned char* mistakes) {
    const unsigned int member = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int member_offset = member * 81;
    for(unsigned int col = 0; col < 9; col++) {
        unsigned char is_illegal = mistakes[member + col];
        unsigned int other = somehow_get_another();

        unsigned char able[9] = {1,1,1, 1,1,1, 1,1,1};
        unsigned int a_spot[9] = {0,0,0 ,0,0,0 ,0,0,0};
        unsigned int b_spot[9] = {0,0,0 ,0,0,0 ,0,0,0};
        unsigned int a_mask = 0;
        unsigned int b_mask = 0;
        unsigned int a_spotted = 0;
        unsigned int b_spotted = 0;

        // find repeated numbers in both columns
        for(unsigned int row = 0; row < 9; row++) {
            unsigned int a_i = member_offset + (row * 9 + col);
            unsigned int b_i = member_offset + (row * 9 + other);
            unsigned int a_val = grid[a_i] - 1;
            unsigned int b_val = grid[b_i] - 1;
            unsigned int a_bin_val = 1 << a_val;
            unsigned int b_bin_val = 1 << b_val;
            unsigned char a_is_hint = occupancy[a_i];
            unsigned char b_is_hint = occupancy[b_i];
            unsigned char a_not_hint = 1 ^ a_is_hint;
            unsigned char b_not_hint = 1 ^ b_is_hint;
            unsigned int a_change = a_spotted & a_bin_val;
            unsigned int b_change = b_spotted & b_bin_val;
            a_mask |= (a_change & (((a_not_hint) << row)  | ((a_not_hint - 1) & a_spot[a_val])));
            b_mask |= (b_change & (((b_not_hint) << row)  | ((b_not_hint - 1) & b_spot[b_val])));
            a_spotted |= ((a_change - 1) & a_bin_val);
            b_spotted |= ((b_change - 1) & b_bin_val);
            a_spot[a_val] = ((a_change - 1) & (1 << row));
            b_spot[b_val] = ((b_change - 1) & (1 << row));
        }

        // change numbers if repeat rows coincide
        unsigned int match_mask = a_mask & b_mask;
        unsigned int row = 0;
        if(match_mask) {
            for (row = 0; row < 9; row++) {
                unsigned int row_offset = row * 9;
                // check validity of changing these numbers
                unsigned int a_num = grid[member_offset + row_offset + col];
                unsigned int b_num = grid[member_offset + row_offset + other];
                unsigned char a_valid = able[a_num];
                unsigned char b_valid = able[b_num];
                unsigned char valid = a_valid & b_valid;
                able[a_num] &= valid;
                able[b_num] &= valid;
                // if a is not in b and vice versa change
                unsigned int a_val = 1 << a_num;
                unsigned int b_val = 1 << b_num;
                unsigned int good_change = (((a_val & b_spotted)  |  (b_val & a_spotted)) - 1) & valid;
                grid[member_offset + row_offset + col] = (good_change & b_num) | (!good_change & a_num);
                grid[member_offset + row_offset + other] = (good_change & a_num) | (!good_change & b_num);
            }
        }
    }
}

int main() {
    // Assuming N is the number of members and is defined
    int N = 1024; // Example value

    // Allocate host memory and initialize data here
    unsigned char *h_grid, *h_occupancy, *h_mistakes;

    // Allocate device memory
    unsigned char *d_grid, *d_occupancy, *d_mistakes;
    hipMalloc(&d_grid, N * 81 * sizeof(unsigned char));
    hipMalloc(&d_occupancy, N * 81 * sizeof(unsigned char));
    hipMalloc(&d_mistakes, N * 9 * sizeof(unsigned char));

    // Copy data from host to device
    hipMemcpy(d_grid, h_grid, N * 81 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_occupancy, h_occupancy, N * 81 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_mistakes, h_mistakes, N * 9 * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    // Launch the kernel
    lsga_col_kernel<<<gridDim, blockDim>>>(d_grid, d_occupancy, d_mistakes);

    // Copy the results back to host
    hipMemcpy(h_grid, d_grid, N * 81 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_grid);
    hipFree(d_occupancy);
    hipFree(d_mistakes);

    // Free host memory and perform other cleanup here

    return 0;
}